#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <opencv2/opencv.hpp>
#include "convolution.h"
#include "helpers.h"
using namespace std;
using namespace cv;
void testConvolution()
{
    cv::Mat img = getRawImage("./Lena.pgm");
    img.convertTo(img, CV_32FC1);
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 2, 2, "global_only", true, "results/kernel2x2_size2x2_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 3, 2, "global_only", true, "results/kernel2x2_size3x3_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 4, 2, "global_only", true, "results/kernel2x2_size4x4_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 2, "global_only", true, "results/kernel2x2_size5x5_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 2, "global_only", true, "results/kernel2x2_size6x6_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 2, "global_only", true, "results/kernel2x2_size7x7_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 8, 2, "global_only", true, "results/kernel2x2_size8x8_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 3, 3, "global_only", true, "results/kernel3x3_size3x3_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 4, 3, "global_only", true, "results/kernel3x3_size4x4_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 3, "global_only", true, "results/kernel3x3_size5x5_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 3, "global_only", true, "results/kernel3x3_size6x6_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 3, "global_only", true, "results/kernel3x3_size7x7_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 8, 3, "global_only", true, "results/kernel3x3_size8x8_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 9, 3, "global_only", true, "results/kernel3x3_size9x9_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 4, 4, "global_only", true, "results/kernel4x4_size4x4_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 4, "global_only", true, "results/kernel4x4_size5x5_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 4, "global_only", true, "results/kernel4x4_size6x6_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 4, "global_only", true, "results/kernel4x4_size7x7_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 8, 4, "global_only", true, "results/kernel4x4_size8x8_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 9, 4, "global_only", true, "results/kernel4x4_size9x9_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 10, 4, "global_only", true, "results/kernel4x4_size10x10_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 5, "global_only", true, "results/kernel5x5_size5x5_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 5, "global_only", true, "results/kernel5x5_size6x6_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 5, "global_only", true, "results/kernel5x5_size7x7_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 8, 5, "global_only", true, "results/kernel5x5_size8x8_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 9, 5, "global_only", true, "results/kernel5x5_size9x9_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 10, 5, "global_only", true, "results/kernel5x5_size10x10_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 11, 5, "global_only", true, "results/kernel5x5_size11x11_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 6, "global_only", true, "results/kernel6x6_size6x6_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 6, "global_only", true, "results/kernel6x6_size7x7_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 8, 6, "global_only", true, "results/kernel6x6_size8x8_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 9, 6, "global_only", true, "results/kernel6x6_size9x9_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 10, 6, "global_only", true, "results/kernel6x6_size10x10_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 11, 6, "global_only", true, "results/kernel6x6_size11x11_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 12, 6, "global_only", true, "results/kernel6x6_size12x12_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 7, "global_only", true, "results/kernel7x7_size7x7_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 8, 7, "global_only", true, "results/kernel7x7_size8x8_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 9, 7, "global_only", true, "results/kernel7x7_size9x9_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 10, 7, "global_only", true, "results/kernel7x7_size10x10_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 11, 7, "global_only", true, "results/kernel7x7_size11x11_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 12, 7, "global_only", true, "results/kernel7x7_size12x12_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 13, 7, "global_only", true, "results/kernel7x7_size13x13_global_only.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 2, 2, "global_register", true, "results/kernel2x2_size2x2_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 3, 2, "global_register", true, "results/kernel2x2_size3x3_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 4, 2, "global_register", true, "results/kernel2x2_size4x4_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 2, "global_register", true, "results/kernel2x2_size5x5_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 2, "global_register", true, "results/kernel2x2_size6x6_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 2, "global_register", true, "results/kernel2x2_size7x7_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 3, 3, "global_register", true, "results/kernel3x3_size3x3_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 4, 3, "global_register", true, "results/kernel3x3_size4x4_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 3, "global_register", true, "results/kernel3x3_size5x5_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 3, "global_register", true, "results/kernel3x3_size6x6_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 3, "global_register", true, "results/kernel3x3_size7x7_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 4, 4, "global_register", true, "results/kernel4x4_size4x4_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 4, "global_register", true, "results/kernel4x4_size5x5_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 4, "global_register", true, "results/kernel4x4_size6x6_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 4, "global_register", true, "results/kernel4x4_size7x7_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 5, 5, "global_register", true, "results/kernel5x5_size5x5_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 5, "global_register", true, "results/kernel5x5_size6x6_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 5, "global_register", true, "results/kernel5x5_size7x7_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 6, 6, "global_register", true, "results/kernel6x6_size6x6_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 6, "global_register", true, "results/kernel6x6_size7x7_global_register.png");
    convolutionWrapper((float*)&img.data[0], img.cols, img.rows, 7, 7, "global_register", true, "results/kernel7x7_size7x7_global_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 2, 2, "texCache_only", true, "results/kernel2x2_size2x2_texCache_only.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 3, 3, "texCache_only", true, "results/kernel3x3_size3x3_texCache_only.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 4, 4, "texCache_only", true, "results/kernel4x4_size4x4_texCache_only.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 5, 5, "texCache_only", true, "results/kernel5x5_size5x5_texCache_only.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 6, 6, "texCache_only", true, "results/kernel6x6_size6x6_texCache_only.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 7, "texCache_only", true, "results/kernel7x7_size7x7_texCache_only.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 2, 2, "texCache_register", true, "results/kernel2x2_size2x2_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 3, 2, "texCache_register", true, "results/kernel2x2_size3x3_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 4, 2, "texCache_register", true, "results/kernel2x2_size4x4_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 5, 2, "texCache_register", true, "results/kernel2x2_size5x5_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 6, 2, "texCache_register", true, "results/kernel2x2_size6x6_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 2, "texCache_register", true, "results/kernel2x2_size7x7_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 3, 3, "texCache_register", true, "results/kernel3x3_size3x3_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 4, 3, "texCache_register", true, "results/kernel3x3_size4x4_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 5, 3, "texCache_register", true, "results/kernel3x3_size5x5_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 6, 3, "texCache_register", true, "results/kernel3x3_size6x6_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 3, "texCache_register", true, "results/kernel3x3_size7x7_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 4, 4, "texCache_register", true, "results/kernel4x4_size4x4_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 5, 4, "texCache_register", true, "results/kernel4x4_size5x5_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 6, 4, "texCache_register", true, "results/kernel4x4_size6x6_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 4, "texCache_register", true, "results/kernel4x4_size7x7_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 5, 5, "texCache_register", true, "results/kernel5x5_size5x5_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 6, 5, "texCache_register", true, "results/kernel5x5_size6x6_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 5, "texCache_register", true, "results/kernel5x5_size7x7_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 6, 6, "texCache_register", true, "results/kernel6x6_size6x6_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 6, "texCache_register", true, "results/kernel6x6_size7x7_texCache_register.png");
    convolutionWrapper_texCache((float*)&img.data[0], img.cols, img.rows, 7, 7, "texCache_register", true, "results/kernel7x7_size7x7_texCache_register.png");
}

void testConvolution_withDummyImg(int height, int width)
{
    float* img = getDummyImg(height, width);
    FILE * pFile = fopen("perf.txt", "w");
    fprintf(pFile, "kernelSize amountToLoad memoryScheme responseTime\n");
    int nRuns = 10;
    float responseTime = 0;
    responseTime = convolutionWrapper(img, width, height, 3, 3, "global_register", false); //warmup
    printf("memoryScheme = %s \n", "global_only");
    for(int kernelSize=2; kernelSize<8; kernelSize++)   
    {
        for(int sqrtConvsPerThread=1; sqrtConvsPerThread<8; sqrtConvsPerThread++)
        {
            int amountToLoad = sqrtConvsPerThread+kernelSize-1; //actually, prefetching nothing in this version
            responseTime = 0;
            for(int i=0; i<nRuns; i++)
            {
                float tmpTime = convolutionWrapper(img, width, height, amountToLoad, kernelSize, "global_only", false);
                responseTime += tmpTime;
            }
            responseTime = responseTime/nRuns;
            fprintf(pFile, "%d, %d, %s, %f \n", kernelSize, amountToLoad, "global_only", responseTime);
            printf("kernelSize = %d. amountToLoad = %d. time per Convolution = %f seconds \n", kernelSize, amountToLoad, responseTime);
            hipDeviceSynchronize();
        }
        printf("\n");
    }
    printf("memoryScheme = %s \n", "global_register");
    for(int kernelSize=2; kernelSize<8; kernelSize++)   
    {
        for(int amountToLoad=kernelSize; amountToLoad<8; amountToLoad++)
        {
            responseTime = 0;
            for(int i=0; i<nRuns; i++)
            {
                float tmpTime = convolutionWrapper(img, width, height, amountToLoad, kernelSize, "global_register", false);
                responseTime += tmpTime;
            }
            responseTime = responseTime/nRuns;
            fprintf(pFile, "%d, %d, %s, %f \n", kernelSize, amountToLoad, "global_register", responseTime);
            printf("kernelSize = %d. amountToLoad = %d. time per Convolution = %f seconds \n", kernelSize, amountToLoad, responseTime);
            hipDeviceSynchronize();
        }
        printf("\n");
    }
    printf("memoryScheme = %s \n", "texCache_only");
    for(int kernelSize=2; kernelSize<8; kernelSize++)   
    {
        int amountToLoad = kernelSize;
        responseTime = 0;
        for(int i=0; i<nRuns; i++)
        {
            float tmpTime = convolutionWrapper_texCache(img, width, height, amountToLoad, kernelSize, "texCache_only", false);
            responseTime += tmpTime;
        }
        responseTime = responseTime/nRuns;
        fprintf(pFile, "%d, %d, %s, %f \n", kernelSize, amountToLoad, "texCache_only", responseTime);
        printf("kernelSize = %d. amountToLoad = %d. time per Convolution = %f seconds \n", kernelSize, amountToLoad, responseTime);
        hipDeviceSynchronize();
        printf("\n");
    }
    printf("memoryScheme = %s \n", "texCache_register");
    for(int kernelSize=2; kernelSize<8; kernelSize++)   
    {
        for(int amountToLoad=kernelSize; amountToLoad<8; amountToLoad++)
        {
            responseTime = 0;
            for(int i=0; i<nRuns; i++)
            {
                float tmpTime = convolutionWrapper_texCache(img, width, height, amountToLoad, kernelSize, "texCache_register", false);
                responseTime += tmpTime;
            }
            responseTime = responseTime/nRuns;
            fprintf(pFile, "%d, %d, %s, %f \n", kernelSize, amountToLoad, "texCache_register", responseTime);
            printf("kernelSize = %d. amountToLoad = %d. time per Convolution = %f seconds \n", kernelSize, amountToLoad, responseTime);
            hipDeviceSynchronize();
        }
        printf("\n");
    }
    fclose(pFile);
}
